#include "hip/hip_runtime.h"

//Ran@2018/3/30

#include <exception>
#include <iostream>
#include <fstream>
#include <string>
#include <random>
#include <cstdio>

#include <hip/hip_runtime.h>

#include "timer.h"

//kernel����
__global__ void vectorAdd(float *a, float *b, float *c, int num)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	c[i] = a[i] + a[i];
}

//������
void errProc(hipError_t err, std::string errStr = "")
{
	if (err != hipSuccess)
	{
		throw std::exception((errStr + hipGetErrorString(err)).c_str());
	}
}

int main()
{
	//������
	constexpr long num = 500000;
	constexpr long size = num * sizeof(float);

	//��ʱ��
	Timer t;
	
	int device;
	hipDeviceProp_t prop;
	hipGetDevice(&device);
	hipGetDeviceProperties(&prop, device);
	std::cout << "Device name: " << prop.name << std::endl;
	std::cout << "Device memory: " << prop.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
	std::cout << "Memory Frequency: " << prop.memoryClockRate / 1000 << "MHz" << std::endl;
	std::cout << "MultiProcessor: " << prop.multiProcessorCount << std::endl;
	std::cout << "Clock rate: " << prop.clockRate / 1000 << "MHz" << std::endl;
	std::cout << "Max threads pre multiprocessor: " << prop.maxThreadsPerMultiProcessor << std::endl;
	std::cout << "Max blocks: x: " << prop.maxGridSize[0] 
		<< " y: "<<prop.maxGridSize[1] 
		<< " z: " << prop.maxGridSize[2] << std::endl;
	std::cout << "Max threads per block: " <<  prop.maxThreadsPerBlock << std::endl;
	std::cout << "Max threads in each dims: x: " << prop.maxThreadsDim[0]
		<< " y: " << prop.maxThreadsDim[1]
		<< " z: " << prop.maxThreadsDim[2] << std::endl;
	std::cout << "Warp size:" << prop.warpSize << std::endl;

	try
	{
		//�ڴ���
		float *ra=new float[num];
		float *rb=new float[num];
		float *rc=new float[num];

		//�Դ���
		float *ga = NULL;
		float *gb = NULL;
		float *gc = NULL;

		//���������
		auto rand = []()
		{
			std::uniform_real_distribution<float> uni(0, 10000);
			return uni(std::random_device());
		};

		//�����������
		for (int i = 0; i != num; ++i)
		{
			ra[i] = rand();
			rb[i] = rand();
		}

		t.begin();

		//���Դ��з���
		errProc(hipMalloc(&ga, size), "����Aʧ��");
		errProc(hipMalloc(&gb, size), "����Bʧ��");
		errProc(hipMalloc(&gc, size), "����Cʧ��");

		//���Ƶ��Դ�
		errProc(hipMemcpy(ga, ra, size, hipMemcpyHostToDevice), "����Aʧ��");
		errProc(hipMemcpy(gb, rb, size, hipMemcpyHostToDevice), "����Bʧ��");

		//����

		//ÿ�����̸߳���
		const int threads = 1024;
		//������
		const int block = (num + threads - 1) / threads;
		vectorAdd << <threads, block>> > (ga, gb, gc, num);
		errProc(hipGetLastError(), "�޷�����");

		//ȡ���
		errProc(hipMemcpy(rc, gc, size, hipMemcpyDeviceToHost), "�޷���ȡ���");
		
		//�ͷ��ڴ�
		hipFree(ga), 
		hipFree(gb);
		hipFree(gc);

		t.end();
		std::cout << "GPU������ɣ���ʱ��" << t.time() << std::endl;

		t.reset();
		t.begin();

		for (int i = 0; i < num; ++i)
		{
			rc[i] = ra[i] + rb[i];
		}

		t.end();
		std::cout << "CPU������ɣ���ʱ��" << t.time() << std::endl;

		delete[] ra;
		delete[] rb;
		delete[] rc;
	}
	catch (std::exception e)
	{
		std::cerr << e.what() << std::endl;
	}
	
	std::cout << "��������˳�" << std::endl;
	getchar();

	return 0;
}
