#include "hip/hip_runtime.h"

//
#include <cstdlib>
#include <exception>
#include <cstdio>
#include <utility>
#include <memory>
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>

#include <hip/hip_runtime.h>

#include "ran_timer.h"
#include "ran_helper_functions.h"

class Matrix;

__global__ void matrixMul(Matrix *a, Matrix *b, Matrix *c);

class Matrix
{
	//�������ڴ�������
	std::unique_ptr<float[]> hostData;
	//�������Դ������
	float *deviceData;
	//�������Դ��еĸ���
	Matrix *deviceMat;
	//��ȣ�ÿ�е�������
	size_t _width;
	//�߶ȣ�����
	size_t _height;
	//������
	size_t _size;

	//��ʼ��CUDA����������
	void initCUDACompution()
	{
		errProc(hipMalloc(&deviceData, size() * sizeof(float)),
			"could not malloc memory deviceData in device");
		errProc(hipMemcpy(deviceData, hostData.get(), size() * sizeof(float), hipMemcpyHostToDevice),
			"could not copy memory from hostData to deviceData");
		
		errProc(hipMalloc(&deviceMat, sizeof(Matrix)));
		errProc(hipMemcpy(deviceMat, this, sizeof(Matrix), hipMemcpyHostToDevice));
	}
	void clearDeviceMem()
	{
		if (deviceData != nullptr)
		{
			hipFree(deviceData);
			deviceData = nullptr;
		}
		if (deviceMat != nullptr)
		{
			hipFree(deviceMat);
			deviceMat = nullptr;
		}
	}
	//����CUDA����
	void endCUDACompution()
	{
		clearDeviceMem();
	}

public:
	//kernel���в���
	static int threads;

	Matrix() :_width(1), _height(1), _size(1), 
		deviceData(nullptr), deviceMat(nullptr)
	{
		hostData = std::make_unique<float[]>(size());
		zeroing();
	}
	//wΪ��ȣ�hΪ�߶�
	Matrix(size_t w, size_t h) :_width(w), _height(h), _size(w*h), 
		deviceData(nullptr), deviceMat(nullptr)
	{
		hostData = std::make_unique<float[]>(size());
		zeroing();
	}
	//
	Matrix(Matrix& m) :_width(m.width()), _height(m.height()), _size(m.size()),
		deviceData(nullptr), deviceMat(nullptr)
	{
		hostData = std::make_unique<float[]>(size());
		for (size_t i = 0; i != size(); ++i)
		{
			hostData[i] = m[i];
		}
	}
	~Matrix()
	{
		//delete[] hostData;
	}
	//����Ŀ�ȣ�������
	size_t width() const
	{
		return _width;
	}
	//����ĸ߶ȣ�������
	size_t height() const
	{
		return _height;
	}
	//�����������
	size_t size() const
	{
		return _size;
	}
	//�������������
	void randomize()
	{
		for (size_t i = 0; i != size(); ++i)
		{
			hostData[i] = rand<float>();
		}
	}
	//�������ݹ���
	void zeroing()
	{
		for (size_t i = 0; i != size(); ++i)
		{
			hostData[i] = 0;
		}
	}
	//ͨ��һά�±��������
	float &operator[](size_t id) const
	{
		if (id > size())
			throw std::exception("id out of range in Matrix::operator[]");
		return hostData[id];
	}
	//ͨ����ά�±��������
	float &operator()(size_t x, size_t y) const
	{
		if (x > _width)
			throw std::exception("x out of range in Matrix::operator()");
		if (y > _height)
			throw std::exception("y out of range in Matrix::operator()");
		return hostData[x + y*_width];
	}
	//�������������
	Matrix &operator=(const Matrix& m)
	{
		_height = m.height();
		_width = m.width();
		_size = m.size();
		threads = m.threads;

		hostData = std::make_unique<float[]>(size());
		for (size_t i = 0; i != size(); ++i)
		{
			hostData[i] = m[i];
		}
		return *this;
	}
	//��ֵ
	Matrix &operator=(Matrix&& m)
	{
		_height = m.height();
		_width = m.width();
		_size = m.size();
		threads = m.threads;

		hostData = std::move(m.hostData);
		return *this;
	}
	//�����Դ��е�����
	__device__ float &accDevice(size_t x, size_t y)
	{
		return deviceData[x + y * _width];
	}
	//����Ŀ��
	__device__ size_t widthDevice() const
	{
		return _width;
	}
	//����ĸ߶�
	__device__ size_t heightDevice() const
	{
		return _width;
	}
	//����˷�
	Matrix operator*(Matrix &m)
	{
		Matrix ret(this->height(), m.width());
		try
		{
			//��״����
			if (width() != m.height())
				throw std::exception("the shape of matrix does not satisfy\
					 the requirement of matrix multiplication in Matrix::operator*()");
			//��ʼ�������Ƶ��Դ�
			initCUDACompution();
			m.initCUDACompution();
			ret.initCUDACompution();

			//ִ��
			matrixMul<<<height(), threads>>>(deviceMat, m.deviceMat, ret.deviceMat);
			errProc(hipGetLastError(), "fail to execute the kernel function in Matrix::operator*()");

			//�����
			errProc(hipMemcpy(ret.hostData.get(), ret.deviceData, 
				ret.size()*sizeof(float), hipMemcpyDeviceToHost),
					"could not read result in device in Matrix::operator*()");

			//����
			endCUDACompution();
			m.endCUDACompution();
			ret.endCUDACompution();
		}
		catch (std::exception e)
		{
			std::cerr << e.what();
			clearDeviceMem();
			m.clearDeviceMem();
			ret.clearDeviceMem();
		}
		return ret;
	}
};
//����˷���C=A*B
//ÿ��block����C�е�һ��
__global__ void matrixMul(Matrix *a, Matrix *b, Matrix *c)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int bdim = blockDim.x;
	//tidƫ������ÿ��ѭ������thread�ĸ���
	int i = 0;
	int w = c->widthDevice();
	int ha = a->heightDevice();
	//�������������������ѭ��
	if (w < bdim)
	{
		goto last;
	}
	//չ��ѭ����ÿ��ѭ���������c�е�bid�е�bdim����
#pragma unroll
	for (; i < w; i+=bdim)
	{
		//�����ʱ����
		float temp = 0;
		//
		int k = i + tid;
		for (int j = 0; j != ha; ++j)
		{
			temp += a->accDevice(bid, j)*b->accDevice(j, k);
		}
		c->accDevice(bid, k) = temp;
	}
last:
	int k = i + tid;
	//���ھ����С���ܲ�Ϊ256������������ʣ�ಿ�ֽ��м���
	if (k < w)
	{
		float temp = 0;
		for (int j = 0; j != ha; ++j)
		{
			temp += a->accDevice(bid, j)*b->accDevice(j, k);
		}
		c->accDevice(bid, k) = temp;
	}

}

Matrix multiplication(Matrix &a, Matrix &b)
{
	Matrix ret(a.height(), b.width());
	if (a.width() != b.height())
		return ret;
	for (int i = 0; i != ret.height(); ++i)
	{
		for (int j = 0; j != ret.width(); ++j)
		{
			float temp = 0;
			for (int k = 0; k != a.width(); ++k)
			{
				temp += a(i, k)*b(k, j);
			}
			ret(i, j) = temp;
		}
	}
	return ret;
}

int Matrix::threads = 32;

int main(int argc, char **argv)
{
	std::vector<size_t> n = { 10,50,100, 200, 300, 400, 500, 1000 };
	std::vector<int> thd = { 32,64,128,256 };
	bool enableCheck = false;

	Timer t;
	std::ofstream res("result.txt");

	char buffer[200];
	
	displayInfo(std::cout);

	sprintf(buffer, "size:n\tcpu\tgpu thd:\t");
	res << buffer;
	for (auto i : thd)
	{
		res << i << '\t';
	}
	res << std::endl;

	for (int i = 0; i != n.size(); ++i)
	{
		std::cout << "data size = "<<n[i] << std::endl;
		res << n[i] << "\t";

		Matrix a(n[i], n[i]);
		Matrix b(n[i], n[i]);
		Matrix c_g, c_c;

		a.randomize();
		b.randomize();

		std::cout << "\tCPU begin" << std::endl;
		t.begin();
		c_c = multiplication(a, b);
		t.end();
		res << t.time() << "\t";
		t.reset();

		std::cout << "\tGPU begin" << std::endl;
		for (int j = 0; j != thd.size(); ++j)
		{
			std::cout << "\t\tthd = " << thd[j] << std::endl;

			Matrix::threads = thd[j];

			t.begin();
			c_g = a*b;
			t.end();
			res << t.time() << "\t";
			t.reset();
		}
		res << std::endl;
		if (enableCheck)
		{
			std::ofstream outa("a.txt");
			std::ofstream outb("b.txt");
			std::ofstream out("err.log");
			for (int i = 0; i != c_c.width(); ++i)
			{
				for (int j = 0; j != c_c.height(); ++j)
				{
					outa << a(i, j) << ' ';
					outb << b(i, j) << ' ';
					if (abs(c_c(i, j) - c_g(i, j) > 1e-6))
					{
						sprintf(buffer, "(%d, %d), c_c:%f, c_g:%f\n", i, j, c_c(i, j), c_g(i, j));
						out << buffer << std::endl;
					}
				}
				outa << std::endl;
				outb << std::endl;
			}
		}
	}

	//std::cout << "Press Enter to exit." << std::flush;
	//getchar();

	return 0;
}
