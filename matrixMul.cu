#include "hip/hip_runtime.h"

//
#include <cstdlib>
#include <exception>
#include <cstdio>
#include <utility>
#include <memory>
#include <iostream>
#include <fstream>
#include <cmath>

#include <hip/hip_runtime.h>

#include "ran_timer.h"
#include "ran_helper_functions.h"

class Matrix;

__global__ void matrixMul(Matrix a, Matrix b, Matrix c);

class Matrix
{
	//�������ڴ�������
	std::unique_ptr<float[]> hostData;
	//�������Դ������
	float *deviceData;
	//��ȣ�ÿ�е�������
	size_t _width;
	//�߶ȣ�����
	size_t _height;
	//������
	size_t _size;

	//��ʼ��CUDA����������
	void initCUDACompution()
	{
		errProc(hipMalloc(&deviceData, size() * sizeof(float)),
			"could not malloc memory deviceData in device");
		errProc(hipMemcpy(deviceData, hostData.get(), size() * sizeof(float), hipMemcpyHostToDevice),
			"could not copy memory from hostData to deviceData");
	}
	void clearDeviceMem()
	{
		if (deviceData != nullptr)
		{
			hipFree(deviceData);
			deviceData = nullptr;
		}
	}
	//����CUDA����
	void endCUDACompution()
	{
		clearDeviceMem();
	}

public:

	Matrix() :_width(1), _height(1), _size(1), 
		deviceData(nullptr)
	{
		hostData = std::make_unique<float[]>(size());
		zeroing();
	}
	//wΪ��ȣ�hΪ�߶�
	Matrix(size_t w, size_t h) :_width(w), _height(h), _size(w*h), 
		deviceData(nullptr)
	{ 
		hostData = std::make_unique<float[]>(size());
		zeroing();
	}
	//
	Matrix(Matrix& m) :_width(m.width()), _height(m.height()), _size(m.size()),
		deviceData(nullptr)
	{
		hostData = std::make_unique<float[]>(size());
		for (size_t i = 0; i != size(); ++i)
		{
			hostData[i] = m[i];
		}
	}
	~Matrix()
	{
		//delete[] hostData;
	}
	//����Ŀ�ȣ�������
	size_t width() const
	{
		return _width;
	}
	//����ĸ߶ȣ�������
	size_t height() const
	{
		return _height;
	}
	//�����������
	size_t size() const
	{
		return _size;
	}
	//�������������
	void randomize()
	{
		for (size_t i = 0; i != size(); ++i)
		{
			hostData[i] = rand<float>();
		}
	}
	//�������ݹ���
	void zeroing()
	{
		for (size_t i = 0; i != size(); ++i)
		{
			hostData[i] = 0;
		}
	}
	//ͨ��һά�±��������
	float &operator[](size_t id) const
	{
		if (id > size())
			throw std::exception("id out of range in Matrix::operator[]");
		return hostData[id];
	}
	//ͨ����ά�±��������
	float &operator()(size_t x, size_t y) const
	{
		if (x > _width)
			throw std::exception("x out of range in Matrix::operator()");
		if (y > _height)
			throw std::exception("y out of range in Matrix::operator()");
		return hostData[x + y*_width];
	}
	//�������������
	Matrix &operator=(const Matrix& m)
	{
		_height = m.height();
		_width = m.width();
		_size = m.size();

		hostData = std::make_unique<float[]>(size());
		for (size_t i = 0; i != size(); ++i)
		{
			hostData[i] = m[i];
		}
		return *this;
	}
	//��ֵ
	Matrix &operator=(Matrix&& m)
	{
		_height = m.height();
		_width = m.width();
		_size = m.size();

		hostData = std::move(m.hostData);
		return *this;
	}
	//�����Դ��е�����
	__device__ float &accDevice(size_t x, size_t y)
	{
		return deviceData[x + y * _width];
	}
	//����Ŀ��
	__device__ size_t widthDevice() const
	{
		return _width;
	}
	//����ĸ߶�
	__device__ size_t heightDevice() const
	{
		return _width;
	}
	//����˷�
	Matrix operator*(Matrix &m)
	{
		Matrix ret(this->height(), m.width());
		try
		{
			//��״����
			if (width() != m.height())
				throw std::exception("the shape of matrix does not satisfy\
					 the requirement of matrix multiplication in Matrix::operator*()");
			//��ʼ�������Ƶ��Դ�
			initCUDACompution();
			m.initCUDACompution();
			ret.initCUDACompution();

			//ִ��
			matrixMul<<<height(), 32>>>(*this, m, ret);
			errProc(hipGetLastError(), "fail to execute the kernel function in Matrix::operator*()");

			//�����
			errProc(hipMemcpy(ret.hostData.get(), ret.deviceData, 
				ret.size()*sizeof(float), hipMemcpyDeviceToHost),
					"could not read result in device in Matrix::operator*()");

			//����
			endCUDACompution();
			m.endCUDACompution();
			ret.endCUDACompution();
		}
		catch (std::exception e)
		{
			std::cerr << e.what();
			clearDeviceMem();
			m.clearDeviceMem();
			ret.clearDeviceMem();
		}
		return ret;
	}
};
//����˷���C=A*B
//ÿ��block����C�е�һ��
__global__ void matrixMul(Matrix a, Matrix b, Matrix c)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int bdim = blockDim.x;
	//tidƫ������ÿ�ε�������thread�ĸ���
	int i = 0;
	int w = c.widthDevice();
	int ha = a.heightDevice();
	//�������������������ѭ��
	if (w < bdim)
	{
		goto last;
	}
	//չ��ѭ��
#pragma unroll
	for (; i != w; i+=bdim)
	{
		//�����ʱ����
		float temp = 0;
		//
		int k = i + tid;
		for (int j = 0; j != ha; ++j)
		{
			temp += a.accDevice(bid, j)*b.accDevice(j, k);
		}
		c.accDevice(bid, k) = temp;
	}
last:
	int k = i + tid;
	//���ھ����С���ܲ�Ϊ256������������ʣ�ಿ�ֽ��м���
	if (k < w)
	{
		float temp = 0;
		for (int j = 0; j != ha; ++j)
		{
			temp += a.accDevice(bid, j)*b.accDevice(j, k);
		}
		c.accDevice(bid, k) = temp;
	}

}

Matrix multiplication(Matrix &a, Matrix &b)
{
	Matrix ret(a.height(), b.width());
	if (a.width() != b.height())
		return ret;
	for (int i = 0; i != ret.height(); ++i)
	{
		for (int j = 0; j != ret.width(); ++j)
		{
			float temp = 0;
			for (int k = 0; k != a.width(); ++k)
			{
				temp += a(i, k)*b(k, j);
			}
			ret(i, j) = temp;
		}
	}
	return ret;
}

int main(int argc, char **argv)
{
	Matrix a(10, 10);
	Matrix b(10, 10);
	Matrix c_g, c_c;

	a.randomize();
	b.randomize();
	c_g = a*b;
	c_c = multiplication(a, b);

	std::ofstream outa("a.txt");
	std::ofstream outb("b.txt");
	std::ofstream out("err.log");
	char buffer[200];
	for (int i = 0; i != c_c.width(); ++i)
	{
		for (int j = 0; j != c_c.height(); ++j)
		{
			outa << a(i, j) << ' ';
			outb << b(i, j) << ' ';
			if (abs(c_c(i, j) - c_g(i, j) > 1e-6))
			{
				sprintf(buffer, "(%d, %d), c_c:%f, c_g:%f\n", i, j, c_c(i, j), c_g(i, j));
				out << buffer << std::flush;
			}
		}
		outa << std::endl;
		outb << std::endl;
	}
	system("pause");
	return 0;
}
